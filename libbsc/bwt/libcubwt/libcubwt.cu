#include "hip/hip_runtime.h"
/*--

This file is a part of libcubwt, a library for CUDA accelerated
suffix array and burrows wheeler transform construction.

   Copyright (c) 2022-2023 Ilya Grebnov <ilya.grebnov@gmail.com>

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

Please see the file LICENSE for full copyright and license details.

--*/

#include "libcubwt.cuh"

#if defined(_MSC_VER) && defined(__INTELLISENSE__)
    #define __launch_bounds__(block_size) /* */
    #define __HIPCC__

    #include <vector_functions.h>
    #include <hip/device_functions.h>
    #include <>
#endif

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <utility>

#if defined(__GNUC__) || defined(__clang__) || defined(__HIPCC__)
    #define RESTRICT __restrict__
#elif defined(_MSC_VER) || defined(__INTEL_COMPILER)
    #define RESTRICT __restrict
#else
    #define RESTRICT /* */
#endif

#ifndef __CUDA_ARCH__
    #define CUDA_DEVICE_ARCH                0
#else
    #define CUDA_DEVICE_ARCH                __CUDA_ARCH__
#endif

#if CUDA_DEVICE_ARCH == 750
    #define CUDA_SM_THREADS                 1024
#elif CUDA_DEVICE_ARCH == 860 || CUDA_DEVICE_ARCH == 870 || CUDA_DEVICE_ARCH == 890
    #define CUDA_SM_THREADS                 1536
#else
    #define CUDA_SM_THREADS                 2048
#endif

#define CUDA_WARP_THREADS                   (32)
#define CUDA_BLOCK_THREADS                  (512)
#define CUDA_DEVICE_PADDING                 (16 * CUDA_BLOCK_THREADS)

typedef struct LIBCUBWT_DEVICE_STORAGE
{
    void *          device_alloc1;
    void *          device_alloc2;
    void *          device_alloc3;
    void *          device_alloc4;
    void *          device_alloc5;
    void *          device_alloc6;
    void *          device_alloc7;

    void *          device_rsort_temp_storage;
    size_t          device_rsort_temp_storage_size;

    void *          device_ssort_temp_storage;
    size_t          device_ssort_temp_storage_size;

    uint8_t *       device_T;
    uint8_t *       device_heads;

    uint32_t *      device_SA;
    uint32_t *      device_ISA;

    uint32_t *      device_keys;
    uint32_t *      device_offsets;

    uint4 *         device_descriptors_large;
    uint4 *         device_descriptors_copy;
    uint2 *         device_descriptors_small;

    uint32_t *      device_temp_SA;
    uint32_t *      device_temp_ISA;
    uint32_t *      device_temp_keys;

    uint64_t *      device_keys_temp_keys;
    uint64_t *      device_offsets_ISA;

    void *          host_pinned_storage;
    size_t          host_pinned_storage_size;

    int64_t         max_length;
    uint32_t        num_unsorted_segments;
    uint32_t        num_unsorted_suffixes;
    bool            is_partial_suffix_array;

    int32_t         device_L2_cache_bits;

    hipStream_t    hip_stream;
} LIBCUBWT_DEVICE_STORAGE;

static int64_t libcubwt_get_error_code(hipError_t status)
{
    return
        status == hipErrorOutOfMemory     ? LIBCUBWT_GPU_NOT_ENOUGH_MEMORY :
        status == cudaErrorDevicesUnavailable   ? LIBCUBWT_GPU_NOT_SUPPORTED :
        status == hipErrorNoDevice             ? LIBCUBWT_GPU_NOT_SUPPORTED :
        LIBCUBWT_GPU_ERROR;
}

static hipError_t libcubwt_cuda_safe_call(const char * filename, int32_t line, hipError_t result, hipError_t status = hipSuccess)
{
#if !defined(NDEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "%s(%d): libcubwt_cuda_safe_call failed %d: '%s'.\n", filename, line, result, hipGetErrorString(result));
        fflush(stderr);
    }
#else
    (void)(filename); (void)(line);
#endif

    return result != hipSuccess ? result : status;
}

template <typename T>
__device__ __forceinline__ T libcubwt_warp_reduce_sum(T value) 
{
#if CUDA_DEVICE_ARCH >= 800
    return __reduce_add_sync((uint32_t)-1, value);
#else

    #pragma unroll
    for (uint32_t mask = CUDA_WARP_THREADS / 2; mask > 0; mask >>= 1)
    {
        value = hipcub::Sum()(value, __shfl_xor_sync((uint32_t)-1, value, mask, CUDA_WARP_THREADS));
    }

    return value;
#endif
}

template <typename T>
__device__ __forceinline__ T libcubwt_warp_reduce_max(T value) 
{
#if CUDA_DEVICE_ARCH >= 800
    return __reduce_max_sync((uint32_t)-1, value);
#else

    #pragma unroll
    for (uint32_t mask = CUDA_WARP_THREADS / 2; mask > 0; mask >>= 1)
    {
        value = hipcub::Max()(value, __shfl_xor_sync((uint32_t)-1, value, mask, CUDA_WARP_THREADS));
    }

    return value;
#endif
}

template <typename T>
__device__ __forceinline__ void libcubwt_delay_or_prevent_hoisting(T delay)
{
#if CUDA_DEVICE_ARCH >= 700
    __nanosleep(delay);
#else
    __threadfence_block(); (void)(delay);
#endif
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_gather_values_uint32_kernel(const uint32_t * device_idx, const uint32_t * RESTRICT device_src, uint32_t * device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 4;

    device_idx += block_index; device_dst += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 4)
    {
        const uint4 indexes = *(uint4 *)(device_idx + threadIdx.x * 4);

        *(uint4 *)(device_dst + threadIdx.x * 4) = make_uint4(
            __ldg(device_src + indexes.x),
            __ldg(device_src + indexes.y),
            __ldg(device_src + indexes.z),
            __ldg(device_src + indexes.w));
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[thread_index] = __ldg(device_src + device_idx[thread_index]);
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_scatter_values_uint32_kernel(const uint32_t * RESTRICT device_idx, const uint32_t * RESTRICT device_src, uint32_t * RESTRICT device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 4;

    device_idx += block_index; device_src += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 4)
    {
        const uint4 indexes = __ldg((uint4 *)(device_idx + threadIdx.x * 4));
        const uint4 values  = __ldg((uint4 *)(device_src + threadIdx.x * 4));

        device_dst[indexes.x] = values.x;
        device_dst[indexes.y] = values.y;
        device_dst[indexes.z] = values.z;
        device_dst[indexes.w] = values.w;
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
        }
    }
}

template <bool source_is_aligned>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_scatter_values_uint8_kernel(const uint32_t * RESTRICT device_idx, const uint8_t * RESTRICT device_src, uint8_t * RESTRICT device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 4;

    device_idx += block_index; device_src += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 4)
    {
        const uint4 indexes = __ldg((uint4 *)(device_idx + threadIdx.x * 4));

        if (source_is_aligned)
        {
            const uchar4 values  = __ldg((uchar4 *)(device_src + threadIdx.x * 4));

            device_dst[indexes.x] = values.x;
            device_dst[indexes.y] = values.y;
            device_dst[indexes.z] = values.z;
            device_dst[indexes.w] = values.w;
        }
        else
        {
            device_dst[indexes.x] = __ldg(device_src + threadIdx.x * 4 + 0);
            device_dst[indexes.y] = __ldg(device_src + threadIdx.x * 4 + 1);
            device_dst[indexes.z] = __ldg(device_src + threadIdx.x * 4 + 2);
            device_dst[indexes.w] = __ldg(device_src + threadIdx.x * 4 + 3);
        }
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_permute_block_values_uint32_kernel(const uint32_t * RESTRICT device_idx, const uint32_t * RESTRICT device_src, uint32_t * RESTRICT device_dst, uint32_t n)
{
    __shared__ __align__(32) uint32_t cache[16 * CUDA_BLOCK_THREADS];

    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 16;

    device_idx += block_index; device_src += block_index; device_dst += block_index; n -= block_index;

    if (n >= CUDA_BLOCK_THREADS * 16)
    {
        {
            const uint32_t * RESTRICT thread_idx   = device_idx + threadIdx.x * 4;
            const uint32_t * RESTRICT thread_src   = device_src + threadIdx.x * 4;
                  uint32_t * RESTRICT thread_cache = cache - block_index;

            #pragma unroll
            for (uint32_t round = 0; round < 4; round += 1)
            {
                const uint4 indexes = __ldg((uint4 *)(thread_idx));
                const uint4 values  = __ldg((uint4 *)(thread_src));

                thread_cache[indexes.x] = values.x;
                thread_cache[indexes.y] = values.y;
                thread_cache[indexes.z] = values.z;
                thread_cache[indexes.w] = values.w;

                thread_idx += 4 * CUDA_BLOCK_THREADS; thread_src += 4 * CUDA_BLOCK_THREADS;
            }
        }

        __syncthreads();

        {
            const uint32_t * RESTRICT thread_cache = cache      + threadIdx.x * 4;
                  uint32_t * RESTRICT thread_dst   = device_dst + threadIdx.x * 4;

            #pragma unroll
            for (uint32_t round = 0; round < 4; round += 1)
            {
                *(uint4 *)(thread_dst) = *(uint4 *)(thread_cache);

                thread_cache += 4 * CUDA_BLOCK_THREADS; thread_dst += 4 * CUDA_BLOCK_THREADS;
            }
        }
    }
    else
    {
        {
            uint32_t * RESTRICT thread_cache = cache - block_index;

            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                thread_cache[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
            }
        }

        __syncthreads();

        {
            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                device_dst[thread_index] = cache[thread_index];
            }
        }
    }
}

template <bool source_is_aligned>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_permute_block_values_uint8_kernel(const uint32_t * RESTRICT device_idx, const uint8_t * RESTRICT device_src, uint8_t * RESTRICT device_dst, uint32_t n)
{
    __shared__ __align__(32) uint8_t cache[64 * CUDA_BLOCK_THREADS];

    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 64;

    device_idx += block_index; device_src += block_index; device_dst += block_index; n -= block_index;

    if (n >= CUDA_BLOCK_THREADS * 64)
    {
        {
            const uint32_t * RESTRICT thread_idx   = device_idx + threadIdx.x * 4;
            const uint8_t  * RESTRICT thread_src   = device_src + threadIdx.x * 4;
                  uint8_t  * RESTRICT thread_cache = cache - block_index;

            #pragma unroll
            for (uint32_t round = 0; round < 16; round += 1)
            {
                const uint4 indexes = __ldg((uint4 *)(thread_idx));

                if (source_is_aligned)
                {
                    const uchar4 values = __ldg((uchar4 *)(thread_src));

                    thread_cache[indexes.x] = values.x;
                    thread_cache[indexes.y] = values.y;
                    thread_cache[indexes.z] = values.z;
                    thread_cache[indexes.w] = values.w;
                }
                else
                {
                    thread_cache[indexes.x] = __ldg(thread_src + 0);
                    thread_cache[indexes.y] = __ldg(thread_src + 1);
                    thread_cache[indexes.z] = __ldg(thread_src + 2);
                    thread_cache[indexes.w] = __ldg(thread_src + 3);
                }

                thread_idx += 4 * CUDA_BLOCK_THREADS; thread_src += 4 * CUDA_BLOCK_THREADS;
            }
        }

        __syncthreads();

        {
            const uint8_t  * RESTRICT thread_cache = cache      + threadIdx.x * 4;
                  uint8_t  * RESTRICT thread_dst   = device_dst + threadIdx.x * 4;

            #pragma unroll
            for (uint32_t round = 0; round < 16; round += 1)
            {
                *(uchar4 *)(thread_dst) = *(uchar4 *)(thread_cache);

                thread_cache += 4 * CUDA_BLOCK_THREADS; thread_dst += 4 * CUDA_BLOCK_THREADS;
            }
        }
    }
    else
    {
        {
            uint8_t * RESTRICT thread_cache = cache - block_index;

            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                thread_cache[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
            }
        }

        __syncthreads();

        {
            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                device_dst[thread_index] = cache[thread_index];
            }
        }
    }
}

static hipError_t libcubwt_gather_scatter_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_src_idx, uint32_t * device_src, uint32_t * device_dst_idx, uint32_t * device_dst, int64_t m, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_src_index_value(device_src_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_dst_index(device_dst_idx, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_src_index_value, db_dst_index,
            (uint32_t)m,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        int64_t n_gather_scatter_blocks = (m + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);

        libcubwt_gather_values_uint32_kernel<<<(uint32_t)n_gather_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_src_index_value.Current(), device_src, db_src_index_value.Current(), (uint32_t)m);

        if (sort_start_bit < sort_end_bit)
        {
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
                storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
                db_dst_index, db_src_index_value,
                (uint32_t)m,
                sort_start_bit, sort_end_bit,
                storage->hip_stream));
        }

        if (status == hipSuccess)
        {
            libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_gather_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_dst_index.Current(), db_src_index_value.Current(), device_dst, (uint32_t)m);
        }
    }

    return status;
}

static hipError_t libcubwt_scatter_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_idx, uint32_t * device_src, uint32_t * device_dst, int64_t m, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_index(device_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_value(device_src, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)m,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        int64_t n_scatter_blocks = (m + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);

        libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)m);
    }

    return status;
}

static hipError_t libcubwt_permute_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_idx, uint32_t * device_src, uint32_t * device_dst, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_index(device_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_value(device_src, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)n,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        if (((int64_t)1 << sort_start_bit) <= CUDA_BLOCK_THREADS * 16)
        {
            int64_t n_permute_blocks = (n + CUDA_BLOCK_THREADS * 16 - 1) / (CUDA_BLOCK_THREADS * 16);

            libcubwt_permute_block_values_uint32_kernel<<<(uint32_t)n_permute_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
        }
        else
        {
            int64_t n_scatter_blocks = (n + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);

            libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
        }
    }

    return status;
}

static hipError_t libcubwt_permute_values_uint8(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_idx, uint8_t * device_src, uint8_t * device_dst, int64_t n, uint32_t * device_temp1, uint8_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_index(device_idx, device_temp1);
    hipcub::DoubleBuffer<uint8_t>  db_value(device_src, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits) ? (sort_end_bit - storage->device_L2_cache_bits + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)n,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        if (((int64_t)1 << sort_start_bit) <= CUDA_BLOCK_THREADS * 64)
        {
            int64_t n_permute_blocks = (n + CUDA_BLOCK_THREADS * 64 - 1) / (CUDA_BLOCK_THREADS * 64);

            if (((uintptr_t)db_value.Current()) % 4 == 0)
            {
                libcubwt_permute_block_values_uint8_kernel<true><<<(uint32_t)n_permute_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
            }
            else
            {
                libcubwt_permute_block_values_uint8_kernel<false><<<(uint32_t)n_permute_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
            }
        }
        else
        {
            int64_t n_scatter_blocks = (n + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);

            if (((uintptr_t)db_value.Current()) % 4 == 0)
            {
                libcubwt_scatter_values_uint8_kernel<true><<<(uint32_t)n_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
            }
            else
            {
                libcubwt_scatter_values_uint8_kernel<false><<<(uint32_t)n_scatter_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
            }
        }
    }

    return status;
}

template <bool extra_sentinel_bits>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_initialize_device_arrays_kernel(const uint8_t * RESTRICT device_T, uint32_t * RESTRICT device_SA, uint64_t * RESTRICT device_keys)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 8;

    {
        uint32_t thread_index = block_index + threadIdx.x * 4; device_SA += thread_index;
        ((uint4 *)device_SA)[0] = make_uint4(thread_index + 0, thread_index + 1, thread_index + 2, thread_index + 3);

        thread_index += CUDA_BLOCK_THREADS * 4; device_SA += CUDA_BLOCK_THREADS * 4;
        ((uint4 *)device_SA)[0] = make_uint4(thread_index + 0, thread_index + 1, thread_index + 2, thread_index + 3);
    }

    __shared__ __align__(32) uint4 prefixes[4 * CUDA_BLOCK_THREADS];

    {
        device_T += block_index + threadIdx.x * 8;

        const uint2 current = __ldg((uint2 *)(device_T + 0));
        const uint2 next    = __ldg((uint2 *)(device_T + 8));

        {
            uint4 * RESTRICT thread_prefixes = prefixes + threadIdx.x * 4;

            thread_prefixes[0] = make_uint4
            (
                __byte_perm(current.y, next.x, 0x0123) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.x, current.y, 0x0123),
                __byte_perm(current.y, next.x, 0x1234) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.x, current.y, 0x1234)
            );

            thread_prefixes[1] = make_uint4
            (
                __byte_perm(current.y, next.x, 0x2345) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.x, current.y, 0x2345),
                __byte_perm(current.y, next.x, 0x3456) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.x, current.y, 0x3456)
            );

            thread_prefixes[2] = make_uint4
            (
                __byte_perm(next.x, next.y, 0x0123) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.y, next.x, 0x0123),
                __byte_perm(next.x, next.y, 0x1234) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.y, next.x, 0x1234)
            );

            thread_prefixes[3] = make_uint4
            (
                __byte_perm(next.x, next.y, 0x2345) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.y, next.x, 0x2345),
                __byte_perm(next.x, next.y, 0x3456) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(current.y, next.x, 0x3456)
            );
        }
    }

    __syncwarp();

    {
        device_keys += block_index;

        uint4 * RESTRICT thread_prefixes = (uint4 *)prefixes     + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);
        uint4 * RESTRICT thread_keys     = (uint4 *)device_keys  + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);

        thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
        thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
        thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
        thread_keys[0] = thread_prefixes[0];
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, 1)
static void libcubwt_set_sentinel_keys_kernel(uint64_t * RESTRICT device_keys_end, uint64_t s0, uint64_t s1, uint64_t s2, uint64_t s3, uint64_t s4, uint64_t s5, uint64_t s6, uint64_t s7)
{
    device_keys_end[-8] = s0;
    device_keys_end[-7] = s1;
    device_keys_end[-6] = s2;
    device_keys_end[-5] = s3;
    device_keys_end[-4] = s4;
    device_keys_end[-3] = s5;
    device_keys_end[-2] = s6;
    device_keys_end[-1] = s7;
}

static hipError_t libcubwt_initialize_device_arrays(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, int64_t n)
{
    hipError_t status = hipSuccess;

    if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_T, T, (size_t)n, hipMemcpyHostToDevice, storage->hip_stream))) == hipSuccess)
    {
        int64_t n_initialize_blocks = 1 + (n / (CUDA_BLOCK_THREADS * 8));

        bool extra_sentinel_bits = (T[n - 2] == 0) && (T[n - 1] == 0);
        if (extra_sentinel_bits)
        {
            libcubwt_initialize_device_arrays_kernel<true><<<(uint32_t)n_initialize_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(storage->device_T, storage->device_SA, storage->device_keys_temp_keys);
        }
        else
        {
            libcubwt_initialize_device_arrays_kernel<false><<<(uint32_t)n_initialize_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(storage->device_T, storage->device_SA, storage->device_keys_temp_keys);
        }

        {
            uint64_t a = T[n - 8], b = T[n - 7], c = T[n - 6], d = T[n - 5], e = T[n - 4], f = T[n - 3], g = T[n - 2], h = T[n - 1];

            uint64_t s0 = (a << 56) | (b << 48) | (c << 40) | (d << 32) | (e << 24) | (f << 16) | (g << 8) | (h << 0) | (extra_sentinel_bits ? 7 : 0);
            uint64_t s1 = (b << 56) | (c << 48) | (d << 40) | (e << 32) | (f << 24) | (g << 16) | (h << 8) | (extra_sentinel_bits ? 6 : 0);
            uint64_t s2 = (c << 56) | (d << 48) | (e << 40) | (f << 32) | (g << 24) | (h << 16) | (extra_sentinel_bits ? 5 : 0);
            uint64_t s3 = (d << 56) | (e << 48) | (f << 40) | (g << 32) | (h << 24) | (extra_sentinel_bits ? 4 : 0);
            uint64_t s4 = (e << 56) | (f << 48) | (g << 40) | (h << 32) | (extra_sentinel_bits ? 3 : 0);
            uint64_t s5 = (f << 56) | (g << 48) | (h << 40) | (extra_sentinel_bits ? 2 : 0);
            uint64_t s6 = (g << 56) | (h << 48) | (extra_sentinel_bits ? 1 : 0);
            uint64_t s7 = (h << 56);

            libcubwt_set_sentinel_keys_kernel<<<1, 1, 0, storage->hip_stream>>>(storage->device_keys_temp_keys + n, s0, s1, s2, s3, s4, s5, s6, s7);
        }

        storage->num_unsorted_segments      = (uint32_t)1;
        storage->num_unsorted_suffixes      = (uint32_t)n;
        storage->is_partial_suffix_array    = false;
    }

    return status;
}

static hipError_t libcubwt_sort_suffixes_by_prefix(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n)
{
    hipcub::DoubleBuffer<uint64_t> db_keys(storage->device_keys_temp_keys, storage->device_offsets_ISA);
    hipcub::DoubleBuffer<uint32_t> db_SA(storage->device_SA, storage->device_temp_SA);

    hipError_t status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
        storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
        db_keys, db_SA,
        (uint32_t)n,
        0, 64,
        storage->hip_stream));

    if (db_keys.selector) 
    { 
        std::swap(storage->device_keys_temp_keys, storage->device_offsets_ISA);

        std::swap(storage->device_keys, storage->device_offsets); 
        std::swap(storage->device_temp_keys, storage->device_ISA);
    }

    if (db_SA.selector)
    {
        std::swap(storage->device_SA, storage->device_temp_SA);
    }

    return status;
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_initialization_kernel(uint64_t * RESTRICT device_keys, uint8_t * RESTRICT device_heads, uint4 * RESTRICT device_descriptors_large, uint2 * RESTRICT device_descriptors_small, uint32_t n)
{
    const uint32_t thread_index = blockIdx.x * CUDA_BLOCK_THREADS + threadIdx.x;

    device_descriptors_large += thread_index;
    device_descriptors_small += thread_index;

    device_descriptors_large[0] = make_uint4(0, 0, 0, 0);
    device_descriptors_small[0] = make_uint2(0, 0);

    if (blockIdx.x == 0)
    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            device_descriptors_large[-CUDA_WARP_THREADS] = make_uint4((uint32_t)-1, 0, 0, 0);
            device_descriptors_small[-CUDA_WARP_THREADS] = make_uint2((uint32_t)-1, 0);
        }

        {
            device_keys += threadIdx.x; device_heads += threadIdx.x;
            uint64_t key = (threadIdx.x % 2 == 0) ? 0 : (uint64_t)-1;

            if (threadIdx.x < 2)
            {
                device_keys [-2] = key;
                device_heads[-2] = 1;
            }

            device_keys += n; device_heads += n;

            device_keys [0 * CUDA_BLOCK_THREADS] = key;
            device_keys [1 * CUDA_BLOCK_THREADS] = key;
            device_keys [2 * CUDA_BLOCK_THREADS] = key;
            device_keys [3 * CUDA_BLOCK_THREADS] = key;
            device_heads[0 * CUDA_BLOCK_THREADS] = 1;
            device_heads[1 * CUDA_BLOCK_THREADS] = 1;
            device_heads[2 * CUDA_BLOCK_THREADS] = 1;
            device_heads[3 * CUDA_BLOCK_THREADS] = 1;
        }
    }
}

template <bool scatter_ranks_directly>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_initiatory_kernel(
    const uint32_t *    RESTRICT device_SA,
    const uint64_t *    RESTRICT device_keys,
    uint8_t *           RESTRICT device_heads,
    uint32_t *          RESTRICT device_ISA,
    uint32_t *          RESTRICT device_offsets_begin,
    uint32_t *          RESTRICT device_offsets_end,
    uint4 *             RESTRICT device_descriptors
)
{
    __shared__ __align__(32) uint2 warp_state[1 + CUDA_WARP_THREADS];

    uint32_t    thread_exclusive_suffix_rank;
    uint32_t    thread_suffix_rank[4];

    uchar4      thread_segment_end_flag;

    uint32_t    thread_exclusive_segment_index;
    uint32_t    thread_segment_index[4];

    {
        __shared__ __align__(32) ulonglong2 cache[1 + 2 * CUDA_BLOCK_THREADS];

        {
            device_keys += blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 2;

            if (threadIdx.x == 0) { cache[0] = __ldg((ulonglong2 *)(device_keys - 2)); }
            cache[1 + threadIdx.x + 0 * CUDA_BLOCK_THREADS] = __ldg((ulonglong2 *)(device_keys + 0 * CUDA_BLOCK_THREADS));
            cache[1 + threadIdx.x + 1 * CUDA_BLOCK_THREADS] = __ldg((ulonglong2 *)(device_keys + 2 * CUDA_BLOCK_THREADS));
        }

        __syncthreads();

        {
            const uint32_t block_index  = blockIdx.x * CUDA_BLOCK_THREADS * 4;
            const uint32_t thread_index = block_index + threadIdx.x * 4;

            ulonglong2 key_a = cache[2 * threadIdx.x + 0];
            ulonglong2 key_b = cache[2 * threadIdx.x + 1];
            ulonglong2 key_c = cache[2 * threadIdx.x + 2];

            uchar4 thread_new_heads = make_uchar4(
                (key_a.y != key_b.x) ? (uint8_t)1 : (uint8_t)0,
                (key_b.x != key_b.y) ? (uint8_t)1 : (uint8_t)0,
                (key_b.y != key_c.x) ? (uint8_t)1 : (uint8_t)0,
                (key_c.x != key_c.y) ? (uint8_t)1 : (uint8_t)0);

            *(uchar4 *)(device_heads + thread_index) = thread_new_heads;

            thread_suffix_rank[0] = (thread_new_heads.x != 0) ? (thread_index + 0) : 0;
            thread_suffix_rank[1] = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_suffix_rank[0];
            thread_suffix_rank[2] = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_suffix_rank[1];
            thread_suffix_rank[3] = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_suffix_rank[2];

            thread_segment_end_flag.x = (thread_new_heads.x != 0) && (key_a.x == key_a.y);
            thread_segment_end_flag.y = (thread_new_heads.y != 0) && (thread_new_heads.x == 0);
            thread_segment_end_flag.z = (thread_new_heads.z != 0) && (thread_new_heads.y == 0);
            thread_segment_end_flag.w = (thread_new_heads.w != 0) && (thread_new_heads.z == 0);

            thread_segment_index[0] = (uint32_t)thread_segment_end_flag.x;
            thread_segment_index[1] = thread_segment_index[0] + (uint32_t)thread_segment_end_flag.y;
            thread_segment_index[2] = thread_segment_index[1] + (uint32_t)thread_segment_end_flag.z;
            thread_segment_index[3] = thread_segment_index[2] + (uint32_t)thread_segment_end_flag.w;
        }
    }

    {
        uint32_t thread_inclusive_suffix_rank;
        uint32_t thread_inclusive_segment_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_BLOCK_THREADS / CUDA_WARP_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_rank[3]  , thread_inclusive_suffix_rank  , thread_exclusive_suffix_rank  , (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_segment_index[3], thread_inclusive_segment_index, thread_exclusive_segment_index, (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state[threadIdx.x / CUDA_WARP_THREADS] = make_uint2(thread_inclusive_suffix_rank, thread_inclusive_segment_index);
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t            block_exclusive_suffix_rank   = 0;
            uint32_t            block_exclusive_segment_index = 0;

            uint32_t            warp_inclusive_suffix_rank;
            uint32_t            warp_inclusive_segment_index;

            {
                typedef hipcub::WarpScan<uint32_t, CUDA_BLOCK_THREADS / CUDA_WARP_THREADS> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint2 warp_inclusive_state = warp_state[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.x, warp_inclusive_suffix_rank  , hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.y, warp_inclusive_segment_index, hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = 1;
                const uint32_t descriptor_status_full_aggregate_ready       = 4;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_partial_aggregate_ready, 0, warp_inclusive_suffix_rank, warp_inclusive_segment_index));
                }

                {
                    uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint4 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane     = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.z      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.z : 0;
                            block_descriptor.w      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.w : 0;
                        }

                        {
                            block_exclusive_suffix_rank      = hipcub::Max()(block_exclusive_suffix_rank  , libcubwt_warp_reduce_max(block_descriptor.z));
                            block_exclusive_segment_index    = hipcub::Sum()(block_exclusive_segment_index, libcubwt_warp_reduce_sum(block_descriptor.w));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_rank      = hipcub::Max()(warp_inclusive_suffix_rank  , block_exclusive_suffix_rank  );
                    warp_inclusive_segment_index    = hipcub::Sum()(warp_inclusive_segment_index, block_exclusive_segment_index);
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_full_aggregate_ready, 0, warp_inclusive_suffix_rank, warp_inclusive_segment_index));
                }
            }

            {
                if (threadIdx.x == 0)
                {
                    warp_state[0] = make_uint2(block_exclusive_suffix_rank, block_exclusive_segment_index);
                }

                warp_state[1 + threadIdx.x] = make_uint2(warp_inclusive_suffix_rank, warp_inclusive_segment_index);
            }
        }

        __syncthreads();
    }

    {
        uint2 warp_exclusive_state              = warp_state[threadIdx.x / CUDA_WARP_THREADS];
        
        thread_exclusive_suffix_rank            = hipcub::Max()(thread_exclusive_suffix_rank  , warp_exclusive_state.x);
        thread_exclusive_segment_index          = hipcub::Sum()(thread_exclusive_segment_index, warp_exclusive_state.y);

        thread_suffix_rank[0]                   = hipcub::Max()(thread_suffix_rank[0], thread_exclusive_suffix_rank);
        thread_suffix_rank[1]                   = hipcub::Max()(thread_suffix_rank[1], thread_exclusive_suffix_rank);
        thread_suffix_rank[2]                   = hipcub::Max()(thread_suffix_rank[2], thread_exclusive_suffix_rank);
        thread_suffix_rank[3]                   = hipcub::Max()(thread_suffix_rank[3], thread_exclusive_suffix_rank);

        thread_segment_index[0]                 = hipcub::Sum()(thread_segment_index[0], thread_exclusive_segment_index);
        thread_segment_index[1]                 = hipcub::Sum()(thread_segment_index[1], thread_exclusive_segment_index);
        thread_segment_index[2]                 = hipcub::Sum()(thread_segment_index[2], thread_exclusive_segment_index);
        thread_segment_index[3]                 = hipcub::Sum()(thread_segment_index[3], thread_exclusive_segment_index);

        const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        if (thread_segment_end_flag.x != 0)     { device_offsets_begin[thread_segment_index[0]] = thread_exclusive_suffix_rank; device_offsets_end[thread_segment_index[0]] = thread_index + 0; }
        if (thread_segment_end_flag.y != 0)     { device_offsets_begin[thread_segment_index[1]] = thread_suffix_rank[0];        device_offsets_end[thread_segment_index[1]] = thread_index + 1; }
        if (thread_segment_end_flag.z != 0)     { device_offsets_begin[thread_segment_index[2]] = thread_suffix_rank[1];        device_offsets_end[thread_segment_index[2]] = thread_index + 2; }
        if (thread_segment_end_flag.w != 0)     { device_offsets_begin[thread_segment_index[3]] = thread_suffix_rank[2];        device_offsets_end[thread_segment_index[3]] = thread_index + 3; }

        if (scatter_ranks_directly)
        {
            const uint4 indexes                 = __ldg((uint4 *)(device_SA + thread_index));

            device_ISA[indexes.x]               = thread_suffix_rank[0];
            device_ISA[indexes.y]               = thread_suffix_rank[1];
            device_ISA[indexes.z]               = thread_suffix_rank[2];
            device_ISA[indexes.w]               = thread_suffix_rank[3];
        }
        else
        {
            *(uint4 *)(device_ISA + thread_index) = make_uint4(thread_suffix_rank[0], thread_suffix_rank[1], thread_suffix_rank[2], thread_suffix_rank[3]);
        }
    }
}

template <bool alternate_block_descriptor_statuses, bool scatter_ranks_directly>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_incremental_kernel(
    const uint32_t *    RESTRICT device_SA,
    const uint32_t *    RESTRICT device_keys,
    uint8_t *           RESTRICT device_heads,
    uint32_t *          RESTRICT device_out_SA,
    uint32_t *          RESTRICT device_out_ISA,
    uint32_t *          RESTRICT device_offsets_begin,
    uint32_t *          RESTRICT device_offsets_end,
    uint4 *             RESTRICT device_descriptors,
    const uint4 *       RESTRICT device_descriptors_copy
)
{
    __shared__ __align__(32) uint4    warp_state1[1 + CUDA_WARP_THREADS];
    __shared__ __align__(32) uint32_t warp_state2[1 + CUDA_WARP_THREADS];

    uint32_t    thread_exclusive_suffix_old_rank;
    uint32_t    thread_suffix_old_rank[4];

    uint32_t    thread_exclusive_suffix_new_rank;
    uint32_t    thread_suffix_new_rank[4];

    uchar4      thread_segment_end_flag;

    uint32_t    thread_exclusive_segment_index;
    uint32_t    thread_segment_index[4];

    uchar4      thread_updated_suffix_flag;

    uint32_t    thread_exclusive_suffix_index;
    uint32_t    thread_suffix_index[4];

    {
        const uint32_t block_index  = blockIdx.x * CUDA_BLOCK_THREADS * 4;
        const uint32_t thread_index = block_index + threadIdx.x * 4;

        device_keys += thread_index; device_heads += thread_index;

        uint2 key_a                 = __ldg((uint2 *)(device_keys - 2));
        uint4 key_b                 = __ldg((uint4 *)(device_keys));
        uchar4 thread_old_heads     = *(uchar4 *)(device_heads);

        uchar4 thread_new_heads = make_uchar4(
            (key_a.y != key_b.x) ? (uint8_t)1 : (uint8_t)thread_old_heads.x,
            (key_b.x != key_b.y) ? (uint8_t)1 : (uint8_t)thread_old_heads.y,
            (key_b.y != key_b.z) ? (uint8_t)1 : (uint8_t)thread_old_heads.z,
            (key_b.z != key_b.w) ? (uint8_t)1 : (uint8_t)thread_old_heads.w);

        *(uchar4 *)(device_heads) = thread_new_heads;

        thread_suffix_old_rank[0] = (thread_old_heads.x != 0) ? (thread_index + 0) : 0;
        thread_suffix_old_rank[1] = (thread_old_heads.y != 0) ? (thread_index + 1) : thread_suffix_old_rank[0];
        thread_suffix_old_rank[2] = (thread_old_heads.z != 0) ? (thread_index + 2) : thread_suffix_old_rank[1];
        thread_suffix_old_rank[3] = (thread_old_heads.w != 0) ? (thread_index + 3) : thread_suffix_old_rank[2];

        thread_suffix_new_rank[0] = (thread_new_heads.x != 0) ? (thread_index + 0) : 0;
        thread_suffix_new_rank[1] = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_suffix_new_rank[0];
        thread_suffix_new_rank[2] = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_suffix_new_rank[1];
        thread_suffix_new_rank[3] = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_suffix_new_rank[2];

        thread_segment_end_flag.x = (thread_new_heads.x != 0) && (key_a.x == key_a.y) && (device_heads[-1] == 0);
        thread_segment_end_flag.y = (thread_new_heads.y != 0) && (thread_new_heads.x == 0);
        thread_segment_end_flag.z = (thread_new_heads.z != 0) && (thread_new_heads.y == 0);
        thread_segment_end_flag.w = (thread_new_heads.w != 0) && (thread_new_heads.z == 0);

        thread_segment_index[0] = (uint32_t)thread_segment_end_flag.x;
        thread_segment_index[1] = thread_segment_index[0] + (uint32_t)thread_segment_end_flag.y;
        thread_segment_index[2] = thread_segment_index[1] + (uint32_t)thread_segment_end_flag.z;
        thread_segment_index[3] = thread_segment_index[2] + (uint32_t)thread_segment_end_flag.w;
    }

    {
        uint32_t thread_inclusive_suffix_old_rank;
        uint32_t thread_inclusive_suffix_new_rank;
        uint32_t thread_inclusive_segment_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_BLOCK_THREADS / CUDA_WARP_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_old_rank[3], thread_inclusive_suffix_old_rank, thread_exclusive_suffix_old_rank, (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_new_rank[3], thread_inclusive_suffix_new_rank, thread_exclusive_suffix_new_rank, (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_segment_index[3]  , thread_inclusive_segment_index  , thread_exclusive_segment_index  , (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state1[threadIdx.x / CUDA_WARP_THREADS] = make_uint4(0, thread_inclusive_suffix_old_rank, thread_inclusive_suffix_new_rank, thread_inclusive_segment_index);
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t            block_exclusive_suffix_old_rank = __ldg((uint32_t *)(device_descriptors_copy + blockIdx.x - 1) + 2);
            uint32_t            block_exclusive_suffix_new_rank = 0;
            uint32_t            block_exclusive_segment_index   = 0;

            uint32_t            warp_inclusive_suffix_old_rank;
            uint32_t            warp_inclusive_suffix_new_rank;
            uint32_t            warp_inclusive_segment_index;

            {
                typedef hipcub::WarpScan<uint32_t, CUDA_BLOCK_THREADS / CUDA_WARP_THREADS> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint4 warp_inclusive_state = warp_state1[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.y, warp_inclusive_suffix_old_rank, hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.z, warp_inclusive_suffix_new_rank, hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.w, warp_inclusive_segment_index  , hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 4 : 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 3 : 1;
                const uint32_t descriptor_status_full_aggregate_ready       = scatter_ranks_directly ? (alternate_block_descriptor_statuses ? 0 : 4) : 2;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_partial_aggregate_ready, 0, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index));
                }

                {
                    uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint4 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane     = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.z      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.z : 0;
                            block_descriptor.w      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.w : 0;
                        }

                        {
                            block_exclusive_suffix_new_rank     = hipcub::Max()(block_exclusive_suffix_new_rank , libcubwt_warp_reduce_max(block_descriptor.z));
                            block_exclusive_segment_index       = hipcub::Sum()(block_exclusive_segment_index   , libcubwt_warp_reduce_sum(block_descriptor.w));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_old_rank  = hipcub::Max()(warp_inclusive_suffix_old_rank, block_exclusive_suffix_old_rank);
                    warp_inclusive_suffix_new_rank  = hipcub::Max()(warp_inclusive_suffix_new_rank, block_exclusive_suffix_new_rank);
                    warp_inclusive_segment_index    = hipcub::Sum()(warp_inclusive_segment_index  , block_exclusive_segment_index  );
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_full_aggregate_ready, 0, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index));
                }
            }

            {
                if (threadIdx.x == 0)
                {
                    warp_state1[0] = make_uint4(0, block_exclusive_suffix_old_rank, block_exclusive_suffix_new_rank, block_exclusive_segment_index);
                }

                warp_state1[1 + threadIdx.x] = make_uint4(0, warp_inclusive_suffix_old_rank, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index);
            }
        }

        __syncthreads();
    }

    {
        uint4 warp_exclusive_state              = warp_state1[threadIdx.x / CUDA_WARP_THREADS];
        
        thread_exclusive_suffix_old_rank        = hipcub::Max()(thread_exclusive_suffix_old_rank, warp_exclusive_state.y);
        thread_exclusive_suffix_new_rank        = hipcub::Max()(thread_exclusive_suffix_new_rank, warp_exclusive_state.z);
        thread_exclusive_segment_index          = hipcub::Sum()(thread_exclusive_segment_index  , warp_exclusive_state.w);

        thread_suffix_old_rank[0]               = hipcub::Max()(thread_suffix_old_rank[0], thread_exclusive_suffix_old_rank);
        thread_suffix_old_rank[1]               = hipcub::Max()(thread_suffix_old_rank[1], thread_exclusive_suffix_old_rank);
        thread_suffix_old_rank[2]               = hipcub::Max()(thread_suffix_old_rank[2], thread_exclusive_suffix_old_rank);
        thread_suffix_old_rank[3]               = hipcub::Max()(thread_suffix_old_rank[3], thread_exclusive_suffix_old_rank);

        thread_suffix_new_rank[0]               = hipcub::Max()(thread_suffix_new_rank[0], thread_exclusive_suffix_new_rank);
        thread_suffix_new_rank[1]               = hipcub::Max()(thread_suffix_new_rank[1], thread_exclusive_suffix_new_rank);
        thread_suffix_new_rank[2]               = hipcub::Max()(thread_suffix_new_rank[2], thread_exclusive_suffix_new_rank);
        thread_suffix_new_rank[3]               = hipcub::Max()(thread_suffix_new_rank[3], thread_exclusive_suffix_new_rank);

        thread_segment_index[0]                 = hipcub::Sum()(thread_segment_index[0], thread_exclusive_segment_index);
        thread_segment_index[1]                 = hipcub::Sum()(thread_segment_index[1], thread_exclusive_segment_index);
        thread_segment_index[2]                 = hipcub::Sum()(thread_segment_index[2], thread_exclusive_segment_index);
        thread_segment_index[3]                 = hipcub::Sum()(thread_segment_index[3], thread_exclusive_segment_index);

        const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        if (thread_segment_end_flag.x != 0)     { device_offsets_begin[thread_segment_index[0]] = thread_exclusive_suffix_new_rank; device_offsets_end[thread_segment_index[0]] = thread_index + 0; }
        if (thread_segment_end_flag.y != 0)     { device_offsets_begin[thread_segment_index[1]] = thread_suffix_new_rank[0];        device_offsets_end[thread_segment_index[1]] = thread_index + 1; }
        if (thread_segment_end_flag.z != 0)     { device_offsets_begin[thread_segment_index[2]] = thread_suffix_new_rank[1];        device_offsets_end[thread_segment_index[2]] = thread_index + 2; }
        if (thread_segment_end_flag.w != 0)     { device_offsets_begin[thread_segment_index[3]] = thread_suffix_new_rank[2];        device_offsets_end[thread_segment_index[3]] = thread_index + 3; }

        if (scatter_ranks_directly)
        {
            const uint4    indexes              = __ldg((uint4 *)(device_SA + thread_index));

            if (thread_suffix_old_rank[0] != thread_suffix_new_rank[0])  { device_out_ISA[indexes.x] = thread_suffix_new_rank[0]; }
            if (thread_suffix_old_rank[1] != thread_suffix_new_rank[1])  { device_out_ISA[indexes.y] = thread_suffix_new_rank[1]; }
            if (thread_suffix_old_rank[2] != thread_suffix_new_rank[2])  { device_out_ISA[indexes.z] = thread_suffix_new_rank[2]; }
            if (thread_suffix_old_rank[3] != thread_suffix_new_rank[3])  { device_out_ISA[indexes.w] = thread_suffix_new_rank[3]; }
        }
        else
        {
            thread_updated_suffix_flag.x        = thread_suffix_old_rank[0] != thread_suffix_new_rank[0];
            thread_updated_suffix_flag.y        = thread_suffix_old_rank[1] != thread_suffix_new_rank[1];
            thread_updated_suffix_flag.z        = thread_suffix_old_rank[2] != thread_suffix_new_rank[2];
            thread_updated_suffix_flag.w        = thread_suffix_old_rank[3] != thread_suffix_new_rank[3];

            thread_suffix_index[0]              = (uint32_t)thread_updated_suffix_flag.x;
            thread_suffix_index[1]              = thread_suffix_index[0] + (uint32_t)thread_updated_suffix_flag.y;
            thread_suffix_index[2]              = thread_suffix_index[1] + (uint32_t)thread_updated_suffix_flag.z;
            thread_suffix_index[3]              = thread_suffix_index[2] + (uint32_t)thread_updated_suffix_flag.w;
        }
    }

    if (!scatter_ranks_directly)
    {
        {
            uint32_t thread_inclusive_suffix_index;

            typedef hipcub::WarpScan<uint32_t> WarpScan;

            __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_BLOCK_THREADS / CUDA_WARP_THREADS];

            WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_index[3], thread_inclusive_suffix_index, thread_exclusive_suffix_index, (uint32_t)0, hipcub::Sum());

            if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
            {
                warp_state2[threadIdx.x / CUDA_WARP_THREADS] = thread_inclusive_suffix_index;
            }

            __syncthreads();
        }

        {
            if (threadIdx.x < CUDA_WARP_THREADS)
            {
                uint32_t            block_exclusive_suffix_index = 0;
                uint32_t            warp_inclusive_suffix_index;

                {
                    typedef hipcub::WarpScan<uint32_t, CUDA_BLOCK_THREADS / CUDA_WARP_THREADS> WarpScan;

                    __shared__ typename WarpScan::TempStorage warp_scan_storage;

                    uint32_t warp_inclusive_state = warp_state2[threadIdx.x];

                    WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state, warp_inclusive_suffix_index, hipcub::Sum());
                }

                {
                    const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 2 : 2;
                    const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 1 : 3;
                    const uint32_t descriptor_status_full_aggregate_ready       = alternate_block_descriptor_statuses ? 0 : 4;

                    if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                    {
                        hipcub::ThreadStore<hipcub::STORE_CG>((uint2 *)(device_descriptors + blockIdx.x), make_uint2(descriptor_status_partial_aggregate_ready, warp_inclusive_suffix_index));
                    }

                    {
                        uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                        int32_t full_aggregate_lane, delay = 8;
                        do
                        {
                            descriptors_lookback -= CUDA_WARP_THREADS;

                            uint2 block_descriptor;
                            do
                            {
                                libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                                block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>((uint2 *)descriptors_lookback);
                            } while (__any_sync((uint32_t)-1, alternate_block_descriptor_statuses 
                                ? ((int32_t )block_descriptor.x >= (int32_t )descriptor_status_aggregate_not_ready)
                                : ((uint32_t)block_descriptor.x <= (uint32_t)descriptor_status_aggregate_not_ready)));

                            delay = 0;

                            {
                                full_aggregate_lane = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                                block_descriptor.y  = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.y : 0;
                            }

                            {
                                block_exclusive_suffix_index = hipcub::Sum()(block_exclusive_suffix_index, libcubwt_warp_reduce_sum(block_descriptor.y));
                            }

                        } while (full_aggregate_lane == -1);

                        warp_inclusive_suffix_index = hipcub::Sum()(warp_inclusive_suffix_index, block_exclusive_suffix_index);
                    }

                    if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                    {
                        hipcub::ThreadStore<hipcub::STORE_CG>((uint2 *)(device_descriptors + blockIdx.x), make_uint2(descriptor_status_full_aggregate_ready, warp_inclusive_suffix_index));
                    }
                }

                {
                    if (threadIdx.x == 0)
                    {
                        warp_state2[0] = block_exclusive_suffix_index;
                    }

                    warp_state2[1 + threadIdx.x] = warp_inclusive_suffix_index;
                }
            }

            __syncthreads();
        }

        {
            if (thread_updated_suffix_flag.x + thread_updated_suffix_flag.y + thread_updated_suffix_flag.z + thread_updated_suffix_flag.w > 0)
            {
                uint32_t warp_exclusive_state           = warp_state2[threadIdx.x / CUDA_WARP_THREADS];
        
                thread_exclusive_suffix_index           = hipcub::Sum()(thread_exclusive_suffix_index, warp_exclusive_state);

                thread_suffix_index[0]                  = hipcub::Sum()(thread_suffix_index[0], thread_exclusive_suffix_index);
                thread_suffix_index[1]                  = hipcub::Sum()(thread_suffix_index[1], thread_exclusive_suffix_index);
                thread_suffix_index[2]                  = hipcub::Sum()(thread_suffix_index[2], thread_exclusive_suffix_index);
                thread_suffix_index[3]                  = hipcub::Sum()(thread_suffix_index[3], thread_exclusive_suffix_index);

                const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;
                const uint4    indexes                  = __ldg((uint4 *)(device_SA + thread_index));

                if (thread_updated_suffix_flag.x != 0)  { device_out_SA[thread_suffix_index[0]] = indexes.x; device_out_ISA[thread_suffix_index[0]] = thread_suffix_new_rank[0]; }
                if (thread_updated_suffix_flag.y != 0)  { device_out_SA[thread_suffix_index[1]] = indexes.y; device_out_ISA[thread_suffix_index[1]] = thread_suffix_new_rank[1]; }
                if (thread_updated_suffix_flag.z != 0)  { device_out_SA[thread_suffix_index[2]] = indexes.z; device_out_ISA[thread_suffix_index[2]] = thread_suffix_new_rank[2]; }
                if (thread_updated_suffix_flag.w != 0)  { device_out_SA[thread_suffix_index[3]] = indexes.w; device_out_ISA[thread_suffix_index[3]] = thread_suffix_new_rank[3]; }
            }
        }
    }
}

static hipError_t libcubwt_rank_and_segment_suffixes(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n, int64_t iteration)
{
    hipError_t status                      = hipSuccess;
    int64_t     n_segmentation_blocks       = 1 + (n / (CUDA_BLOCK_THREADS * 4));
    int64_t     n_initialization_blocks     = (n_segmentation_blocks + CUDA_BLOCK_THREADS - 1) / CUDA_BLOCK_THREADS;
    bool        scatter_ranks_directly      = (n <= ((int64_t)1 << (storage->device_L2_cache_bits - 3)));

    if (iteration == 0)
    {
        libcubwt_rank_and_segment_suffixes_initialization_kernel<<<(uint32_t)n_initialization_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
            storage->device_keys_temp_keys,
            storage->device_heads,
            storage->device_descriptors_large,
            storage->device_descriptors_small,
            (uint32_t)n);

        if (scatter_ranks_directly)
        {
            libcubwt_rank_and_segment_suffixes_initiatory_kernel<true><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                storage->device_SA,
                storage->device_keys_temp_keys,
                storage->device_heads,
                storage->device_ISA,
                storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                storage->device_descriptors_large);
        }
        else
        {
            libcubwt_rank_and_segment_suffixes_initiatory_kernel<false><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                NULL,
                storage->device_keys_temp_keys,
                storage->device_heads,
                storage->device_temp_ISA,
                storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                storage->device_descriptors_large);
        }

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_large[n_segmentation_blocks - 1], sizeof(uint4), hipMemcpyDeviceToHost, storage->hip_stream));
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

        if (status == hipSuccess)
        {
            storage->num_unsorted_segments = ((uint4 *)storage->host_pinned_storage)->w;

            if (!scatter_ranks_directly)
            {
                if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_temp_SA, storage->device_SA, n * sizeof(uint32_t), hipMemcpyDeviceToDevice, storage->hip_stream))) == hipSuccess)
                {
                    status = libcubwt_permute_values_uint32(storage, storage->device_temp_SA, storage->device_temp_ISA, storage->device_ISA, n, storage->device_keys, storage->device_temp_keys);
                }
            }
        }
    }
    else
    {
        if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_descriptors_copy - 1, storage->device_descriptors_large - 1, n_segmentation_blocks * sizeof(uint4), hipMemcpyDeviceToDevice, storage->hip_stream))) == hipSuccess)
        {
            if (scatter_ranks_directly)
            {
                if ((iteration % 2) == 0)
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<false, true><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        NULL, storage->device_ISA,
                        storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
                else
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<true, true><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        NULL, storage->device_ISA,
                        storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
            }
            else
            {
                if ((iteration % 2) == 0)
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<false, false><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        storage->device_temp_SA - 1, storage->device_temp_ISA - 1,
                        storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
                else
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<true, false><<<(uint32_t)n_segmentation_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        storage->device_temp_SA - 1, storage->device_temp_ISA - 1,
                        storage->device_offsets - 1, storage->device_offsets + (storage->max_length / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
            }

            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_large[n_segmentation_blocks - 1], sizeof(uint4), hipMemcpyDeviceToHost, storage->hip_stream));
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

            if (status == hipSuccess)
            {
                storage->num_unsorted_segments = ((uint4 *)storage->host_pinned_storage)->w;

                if (!scatter_ranks_directly)
                {
                    uint32_t num_updated_suffixes = ((uint4 *)storage->host_pinned_storage)->y;

                    if (num_updated_suffixes > 0)
                    {
                        status = libcubwt_scatter_values_uint32(storage, storage->device_temp_SA, storage->device_temp_ISA, storage->device_ISA, num_updated_suffixes, n, storage->device_keys, storage->device_temp_keys);
                    }
                }
            }
        }
    }

    return status;
}

template <bool alternate_block_descriptor_statuses>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_gather_unsorted_suffixes_kernel(
    const uint8_t *     RESTRICT device_heads, 
    const uint32_t *    RESTRICT device_SA,
    uint32_t *          RESTRICT device_out_keys,
    uint32_t *          RESTRICT device_out_SA,
    uint2 *             RESTRICT device_descriptors)
{
    __shared__ __align__(32) uint32_t warp_state[1 + CUDA_WARP_THREADS];

    uchar4      thread_updated_suffix_flag;

    uint32_t    thread_exclusive_suffix_index;
    uint32_t    thread_suffix_index[4];

    {
        device_heads += blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        const uchar4    current_heads   = __ldg((uchar4 *)(device_heads));
        const uint8_t   next_head       = current_heads.w > 0 ? __ldg(device_heads + 4) : 0;

        thread_updated_suffix_flag.x    = (current_heads.x + current_heads.y < 2);
        thread_updated_suffix_flag.y    = (current_heads.y + current_heads.z < 2);
        thread_updated_suffix_flag.z    = (current_heads.z + current_heads.w < 2);
        thread_updated_suffix_flag.w    = (current_heads.w +       next_head < 2);

        thread_suffix_index[0]          = (uint32_t)thread_updated_suffix_flag.x;
        thread_suffix_index[1]          = thread_suffix_index[0] + (uint32_t)thread_updated_suffix_flag.y;
        thread_suffix_index[2]          = thread_suffix_index[1] + (uint32_t)thread_updated_suffix_flag.z;
        thread_suffix_index[3]          = thread_suffix_index[2] + (uint32_t)thread_updated_suffix_flag.w;
    }

    {
        uint32_t thread_inclusive_suffix_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_BLOCK_THREADS / CUDA_WARP_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_index[3], thread_inclusive_suffix_index, thread_exclusive_suffix_index, (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state[threadIdx.x / CUDA_WARP_THREADS] = thread_inclusive_suffix_index;
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t block_exclusive_suffix_index = 0;
            uint32_t warp_inclusive_suffix_index;

            {
                typedef hipcub::WarpScan<uint32_t, CUDA_BLOCK_THREADS / CUDA_WARP_THREADS> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint32_t warp_inclusive_state = warp_state[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state, warp_inclusive_suffix_index, hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 2 : 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 1 : 1;
                const uint32_t descriptor_status_full_aggregate_ready       = alternate_block_descriptor_statuses ? 0 : 2;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint2(descriptor_status_partial_aggregate_ready, warp_inclusive_suffix_index));
                }

                {
                    uint2 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint2 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.y  = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.y : 0;
                        }

                        {
                            block_exclusive_suffix_index = hipcub::Sum()(block_exclusive_suffix_index, libcubwt_warp_reduce_sum(block_descriptor.y));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_index = hipcub::Sum()(warp_inclusive_suffix_index, block_exclusive_suffix_index);
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint2(descriptor_status_full_aggregate_ready, warp_inclusive_suffix_index));
                }
            }

            {
                if (threadIdx.x == 0)
                {
                    warp_state[0] = block_exclusive_suffix_index;
                }

                warp_state[1 + threadIdx.x] = warp_inclusive_suffix_index;
            }
        }

        __syncthreads();
    }

    {
        if (thread_updated_suffix_flag.x + thread_updated_suffix_flag.y + thread_updated_suffix_flag.z + thread_updated_suffix_flag.w > 0)
        {
            uint32_t warp_exclusive_state           = warp_state[threadIdx.x / CUDA_WARP_THREADS];
        
            thread_exclusive_suffix_index           = hipcub::Sum()(thread_exclusive_suffix_index, warp_exclusive_state);

            thread_suffix_index[0]                  = hipcub::Sum()(thread_suffix_index[0], thread_exclusive_suffix_index);
            thread_suffix_index[1]                  = hipcub::Sum()(thread_suffix_index[1], thread_exclusive_suffix_index);
            thread_suffix_index[2]                  = hipcub::Sum()(thread_suffix_index[2], thread_exclusive_suffix_index);
            thread_suffix_index[3]                  = hipcub::Sum()(thread_suffix_index[3], thread_exclusive_suffix_index);

            const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;
            const uint4    indexes                  = __ldg((uint4 *)(device_SA + thread_index));

            if (thread_updated_suffix_flag.x != 0)  { device_out_keys[thread_suffix_index[0]] = thread_index + 0; device_out_SA[thread_suffix_index[0]] = indexes.x; }
            if (thread_updated_suffix_flag.y != 0)  { device_out_keys[thread_suffix_index[1]] = thread_index + 1; device_out_SA[thread_suffix_index[1]] = indexes.y; }
            if (thread_updated_suffix_flag.z != 0)  { device_out_keys[thread_suffix_index[2]] = thread_index + 2; device_out_SA[thread_suffix_index[2]] = indexes.z; }
            if (thread_updated_suffix_flag.w != 0)  { device_out_keys[thread_suffix_index[3]] = thread_index + 3; device_out_SA[thread_suffix_index[3]] = indexes.w; }
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_update_suffix_sorting_keys_kernel(const uint8_t * RESTRICT device_heads, const uint32_t * RESTRICT device_SA, const uint32_t * RESTRICT device_ISA, uint32_t * RESTRICT device_keys)
{
    const uint32_t  thread_index    = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

    device_heads += thread_index;

    const uchar4    current_heads   = __ldg((uchar4 *)(device_heads));
    const uint8_t   next_head       = current_heads.w > 0 ? __ldg(device_heads + 4) : 0;

    if (current_heads.x + current_heads.y + current_heads.z + current_heads.w + next_head < 5)
    {
        device_SA += thread_index; device_keys += thread_index;

        const uint4 current_SA = __ldg((uint4 *)(device_SA));

        ((uint4 *)device_keys)[0] = make_uint4(
            (current_heads.x + current_heads.y < 2) ? __ldg(device_ISA + current_SA.x) : (uint32_t)-1,
            (current_heads.y + current_heads.z < 2) ? __ldg(device_ISA + current_SA.y) : (uint32_t)-2,
            (current_heads.z + current_heads.w < 2) ? __ldg(device_ISA + current_SA.z) : (uint32_t)-3,
            (current_heads.w +       next_head < 2) ? __ldg(device_ISA + current_SA.w) : (uint32_t)-4);
    }
}

static hipError_t libcubwt_update_suffix_sorting_keys(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n, int64_t iteration, int64_t depth)
{
    hipError_t status                  = hipSuccess;
    int64_t     n_blocks                = (n + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);
    bool        gather_keys_directly    = (n <= ((int64_t)1 << (storage->device_L2_cache_bits - 2))) || (n > ((int64_t)1 << (storage->device_L2_cache_bits - 2 + 8)));

    if (gather_keys_directly || (storage->num_unsorted_suffixes <= (n / 4)))
    {
        libcubwt_update_suffix_sorting_keys_kernel<<<(uint32_t)n_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(storage->device_heads, storage->device_SA, storage->device_ISA + depth, storage->device_keys);
    }
    else
    {
        if ((iteration % 2) == 0)
        {
            libcubwt_gather_unsorted_suffixes_kernel<false><<<(uint32_t)n_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                storage->device_heads,
                storage->device_SA,
                storage->device_temp_keys - 1, storage->device_temp_SA - 1,
                storage->device_descriptors_small);
        }
        else
        {
            libcubwt_gather_unsorted_suffixes_kernel<true><<<(uint32_t)n_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(
                storage->device_heads,
                storage->device_SA,
                storage->device_temp_keys - 1, storage->device_temp_SA - 1,
                storage->device_descriptors_small);
        }

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_small[n_blocks - 1], sizeof(uint2), hipMemcpyDeviceToHost, storage->hip_stream));
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

        if (status == hipSuccess)
        {
            storage->num_unsorted_suffixes = ((uint2 *)storage->host_pinned_storage)->y;

            if (storage->num_unsorted_suffixes > 0)
            {
                status = libcubwt_gather_scatter_values_uint32(storage, storage->device_temp_SA, storage->device_ISA + depth, storage->device_temp_keys, storage->device_keys, storage->num_unsorted_suffixes, n, storage->device_temp_ISA, storage->device_keys);
            }
        }
    }

    return status;
}

static hipError_t libcubwt_sort_segmented_suffixes_by_rank(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n)
{
    hipcub::DoubleBuffer<uint32_t> d_keys(storage->device_keys, storage->device_temp_keys);
    hipcub::DoubleBuffer<uint32_t> d_values(storage->device_SA, storage->device_temp_SA);

    hipError_t status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceSegmentedSort::SortPairs(
        storage->device_ssort_temp_storage, storage->device_ssort_temp_storage_size,
        d_keys, d_values,
        (int)n, (int)storage->num_unsorted_segments,
        storage->device_offsets, storage->device_offsets + (storage->max_length / 2),
        storage->hip_stream));

    if (d_keys.selector) { std::swap(storage->device_keys, storage->device_temp_keys); }
    if (d_values.selector) { std::swap(storage->device_SA, storage->device_temp_SA); storage->is_partial_suffix_array = true; }

    return status;
}

static hipError_t libcubwt_compute_inverse_suffix_array(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, int64_t n)
{
    hipError_t status = hipSuccess;

    if ((status = libcubwt_initialize_device_arrays(storage, T, n)) == hipSuccess &&
        (status = libcubwt_sort_suffixes_by_prefix(storage, n)) == hipSuccess)
    {
        for (int64_t iteration = 0, depth = 7; true; iteration += 1, depth *= 2)
        {
            if ((status = libcubwt_rank_and_segment_suffixes(storage, n, iteration)) != hipSuccess)
            {
                break;
            }

            if (storage->num_unsorted_segments == 0)
            {
                break;
            }

            if ((status = libcubwt_update_suffix_sorting_keys(storage, n, iteration, depth)) != hipSuccess)
            {
                break;
            }

            if ((status = libcubwt_sort_segmented_suffixes_by_rank(storage, n)) != hipSuccess)
            {
                break;
            }
        }
    }

    return status;
}

static hipError_t libcubwt_copy_suffix_array(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * SA, int64_t n)
{
    return libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(SA, storage->device_SA, n * sizeof(uint32_t), hipMemcpyDeviceToHost, storage->hip_stream));
}

static hipError_t libcubwt_copy_inverse_suffix_array(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * ISA, int64_t n)
{
    return libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(ISA, storage->device_ISA, n * sizeof(uint32_t), hipMemcpyDeviceToHost, storage->hip_stream));
}

static hipError_t libcubwt_synchronize_cuda_stream(LIBCUBWT_DEVICE_STORAGE * storage)
{
    return libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream));
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, 1)
static void libcubwt_extract_auxiliary_indexes_kernel(const uint32_t * RESTRICT device_ISA, uint32_t * RESTRICT device_offsets, uint32_t n, uint32_t r)
{
    for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
    {
        device_offsets[thread_index] = __ldg(device_ISA + (thread_index * r));
    }
}

static hipError_t libcubwt_extract_auxiliary_indexes(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n, int64_t r, uint32_t * I)
{
    hipError_t status      = hipSuccess;
    int64_t     n_indexes   = (n + r - 1) / r;
    uint32_t *  buffer      = ((sizeof(uint32_t) * n_indexes) <= storage->host_pinned_storage_size) ? (uint32_t *)storage->host_pinned_storage : I;

    if (n_indexes > 1)
    {
        libcubwt_extract_auxiliary_indexes_kernel<<<1, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(storage->device_ISA, storage->device_offsets, (uint32_t)n_indexes, (uint32_t)r);

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(buffer, storage->device_offsets, sizeof(uint32_t) * n_indexes, hipMemcpyDeviceToHost, storage->hip_stream), status);
    }
    else
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(buffer, storage->device_ISA, sizeof(uint32_t) * n_indexes, hipMemcpyDeviceToHost, storage->hip_stream), status);
    }

    if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status)) == hipSuccess)
    {
        if (I != buffer) { memcpy(I, buffer, sizeof(uint32_t) * n_indexes); }

        for (int64_t index = 0; index < n_indexes; index += 1) { I[index] += 1; }
    }

    return status;
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_reinitialize_suffix_array_kernel(uint32_t * RESTRICT device_SA)
{
    const uint32_t thread_index = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

    *(uint4 *)(device_SA + thread_index) = make_uint4(thread_index + 0, thread_index + 1, thread_index + 2, thread_index + 3);
}

static hipError_t libcubwt_compute_suffix_array(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n)
{
    hipError_t status = hipSuccess;

    if (storage->is_partial_suffix_array)
    {
        int64_t n_reinitialize_blocks = (n + CUDA_BLOCK_THREADS * 4 - 1) / (CUDA_BLOCK_THREADS * 4);

        libcubwt_reinitialize_suffix_array_kernel<<<(uint32_t)n_reinitialize_blocks, CUDA_BLOCK_THREADS, 0, storage->hip_stream>>>(storage->device_temp_SA);

        status = libcubwt_permute_values_uint32(storage, storage->device_ISA, storage->device_temp_SA, storage->device_SA, n, storage->device_keys, storage->device_temp_keys);
    }

    return status;
}

static hipError_t libcubwt_compute_burrows_wheeler_transform(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, uint8_t * L, int64_t n, int64_t index)
{
    hipError_t status = hipSuccess;

    if ((status = libcubwt_permute_values_uint8(storage, storage->device_ISA, storage->device_T - 1, storage->device_heads, n, storage->device_keys, (uint8_t *)storage->device_temp_keys)) == hipSuccess)
    {
        L[0] = T[n - 1];

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(L + 1, storage->device_heads, (size_t)(index - 1), hipMemcpyDeviceToHost, storage->hip_stream), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(L + index, storage->device_heads + index, (size_t)(n - index), hipMemcpyDeviceToHost, storage->hip_stream), status);
    }

    return status;
}

int64_t libcubwt_allocate_device_storage(void ** device_storage, int64_t max_length)
{
    max_length = ((max_length + (int64_t)(CUDA_DEVICE_PADDING) - (int64_t)1) / (int64_t)(CUDA_DEVICE_PADDING)) * (int64_t)(CUDA_DEVICE_PADDING);

    if ((device_storage == NULL) || (max_length >= UINT32_MAX))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    *device_storage = NULL;

    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)malloc(sizeof(LIBCUBWT_DEVICE_STORAGE));
    if (storage != NULL)
    {
        memset(storage, 0, sizeof(LIBCUBWT_DEVICE_STORAGE));

        hipError_t status = hipSuccess;

        int32_t cuda_device_ordinal;
        int32_t cuda_device_L2_cache_size;

        libcubwt_cuda_safe_call(__FILE__, __LINE__, hipGetDevice(&cuda_device_ordinal), status);
        libcubwt_cuda_safe_call(__FILE__, __LINE__, hipDeviceGetAttribute(&cuda_device_L2_cache_size, hipDeviceAttributeL2CacheSize, cuda_device_ordinal), status);
               
        if (status == hipSuccess)
        {
            {
                storage->device_L2_cache_bits = 0; while (cuda_device_L2_cache_size >>= 1) { storage->device_L2_cache_bits += 1; };
            }

            {
                hipcub::DoubleBuffer<uint8_t> uint8_db;
                hipcub::DoubleBuffer<uint32_t> uint32_db;
                hipcub::DoubleBuffer<uint64_t> uint64_db;

                size_t temp_radix_sort_k32v8 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k32v8, uint32_db, uint8_db, (uint32_t)max_length), status);

                size_t temp_radix_sort_k32v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k32v32, uint32_db, uint32_db, (uint32_t)max_length), status);

                size_t temp_radix_sort_k64v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k64v32, uint64_db, uint32_db, (uint32_t)max_length), status);

                storage->device_rsort_temp_storage_size = std::max(std::max(temp_radix_sort_k32v8, temp_radix_sort_k32v32), temp_radix_sort_k64v32);
            }

            {
                hipcub::DoubleBuffer<uint32_t> uint32_db;

                size_t temp_radix_segmented_sort_k32v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceSegmentedSort::SortPairs(NULL, temp_radix_segmented_sort_k32v32, uint32_db, uint32_db, (int)max_length, (int)max_length / 2, uint32_db.Current(), uint32_db.Current()), status);

                storage->device_ssort_temp_storage_size = std::max(temp_radix_segmented_sort_k32v32, (max_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint32_t));
            }

            int64_t n_descriptors = (max_length / (CUDA_BLOCK_THREADS * 4) + 4) & (-4);

            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc1, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t)), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc2, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t)), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc3, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint32_t)), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc4, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint32_t)), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_ssort_temp_storage, storage->device_ssort_temp_storage_size), status);

            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc5, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t)) , status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc6, (max_length        + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t)) , status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size), status);

            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_alloc7, (3 * n_descriptors + (int64_t)6 * CUDA_DEVICE_PADDING) * sizeof(uint4))   , status);
            
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipHostMalloc((void **)&storage->host_pinned_storage, storage->host_pinned_storage_size = CUDA_DEVICE_PADDING * sizeof(uint32_t)), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamCreate(&storage->hip_stream), status);

            if (status == hipSuccess)
            {
                storage->max_length                 = max_length;

                storage->device_T                   = (uint8_t  *)storage->device_alloc5 + CUDA_DEVICE_PADDING;
                storage->device_heads               = (uint8_t  *)storage->device_alloc6 + CUDA_DEVICE_PADDING;

                storage->device_SA                  = (uint32_t *)storage->device_alloc3 + CUDA_DEVICE_PADDING;
                storage->device_temp_SA             = (uint32_t *)storage->device_alloc4 + CUDA_DEVICE_PADDING;

                storage->device_temp_ISA            = (uint32_t *)storage->device_ssort_temp_storage + CUDA_DEVICE_PADDING;

                storage->device_keys                = (uint32_t *)storage->device_alloc1 + CUDA_DEVICE_PADDING;
                storage->device_offsets             = (uint32_t *)storage->device_alloc2 + CUDA_DEVICE_PADDING;

                storage->device_temp_keys           = (uint32_t *)storage->device_alloc1 + 3 * CUDA_DEVICE_PADDING + max_length;
                storage->device_ISA                 = (uint32_t *)storage->device_alloc2 + 3 * CUDA_DEVICE_PADDING + max_length;

                storage->device_keys_temp_keys      = (uint64_t *)storage->device_alloc1 + CUDA_DEVICE_PADDING;
                storage->device_offsets_ISA         = (uint64_t *)storage->device_alloc2 + CUDA_DEVICE_PADDING;

                storage->device_descriptors_large   = (uint4    *)storage->device_alloc7 + CUDA_DEVICE_PADDING;
                storage->device_descriptors_copy    = (uint4    *)storage->device_alloc7 + 3  * CUDA_DEVICE_PADDING + 1 * n_descriptors;
                storage->device_descriptors_small   = (uint2    *)storage->device_alloc7 + 10 * CUDA_DEVICE_PADDING + 4 * n_descriptors;

                *device_storage = storage;
                return LIBCUBWT_NO_ERROR;
            }
        }

        libcubwt_free_device_storage(storage);

        return libcubwt_get_error_code(status);
    }

    return LIBCUBWT_NOT_ENOUGH_MEMORY;
}

int64_t libcubwt_free_device_storage(void * device_storage)
{
    hipError_t status = hipSuccess;

    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;
    if (storage != NULL)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamDestroy(storage->hip_stream), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipHostFree((void *)storage->host_pinned_storage), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc7), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_rsort_temp_storage), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc6), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc5), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_ssort_temp_storage), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc4), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc3), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc2), status);
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_alloc1), status);

        free(storage);
    }

    return status != hipSuccess ? libcubwt_get_error_code(status) : LIBCUBWT_NO_ERROR;
}

int64_t libcubwt_sa(void * device_storage, const uint8_t * T, uint32_t * SA, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (SA == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_compute_suffix_array(storage, n)) == hipSuccess &&
        (status = libcubwt_copy_suffix_array(storage, SA, n)) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return LIBCUBWT_NO_ERROR;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_isa(void * device_storage, const uint8_t * T, uint32_t * ISA, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (ISA == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_copy_inverse_suffix_array(storage, ISA, n)) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return LIBCUBWT_NO_ERROR;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_sa_isa(void * device_storage, const uint8_t * T, uint32_t * SA, uint32_t * ISA, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (ISA == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_copy_inverse_suffix_array(storage, ISA, n)) == hipSuccess &&
        (status = libcubwt_compute_suffix_array(storage, n)) == hipSuccess &&
        (status = libcubwt_copy_suffix_array(storage, SA, n)) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return LIBCUBWT_NO_ERROR;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_bwt(void * device_storage, const uint8_t * T, uint8_t * L, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (L == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status; uint32_t index;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_extract_auxiliary_indexes(storage, n, n, &index)) == hipSuccess &&
        (status = libcubwt_compute_burrows_wheeler_transform(storage, T, L, n, index)) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return index;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_bwt_isa(void * device_storage, const uint8_t * T, uint8_t * L, uint32_t * ISA, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (L == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status; uint32_t index;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_copy_inverse_suffix_array(storage, ISA, n)) == hipSuccess &&
        (status = libcubwt_extract_auxiliary_indexes(storage, n, n, &index)) == hipSuccess &&
        (status = libcubwt_compute_burrows_wheeler_transform(storage, T, L, n, index)) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return index;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_bwt_aux(void * device_storage, const uint8_t * T, uint8_t * L, int64_t n, int64_t r, uint32_t * I)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (L == NULL) || (n < 16) || (n > storage->max_length) || (r < 2) || ((r & (r - 1)) != 0) || (I == NULL))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status;
    if ((status = libcubwt_compute_inverse_suffix_array(storage, T, n)) == hipSuccess &&
        (status = libcubwt_extract_auxiliary_indexes(storage, n, r, I)) == hipSuccess &&
        (status = libcubwt_compute_burrows_wheeler_transform(storage, T, L, n, I[0])) == hipSuccess &&
        (status = libcubwt_synchronize_cuda_stream(storage)) == hipSuccess)
    {
        return LIBCUBWT_NO_ERROR;
    }

    return libcubwt_get_error_code(status);
}
